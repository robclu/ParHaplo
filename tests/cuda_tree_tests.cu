// ----------------------------------------------------------------------------------------------------------
/// @file   subblock_tests.cpp
/// @brief  Test suite for parahaplo sub-block tests
// ----------------------------------------------------------------------------------------------------------

#define BOOST_TEST_DYN_LINK
#ifdef STAND_ALONE
    #define BOOST_TEST_MODULE SubBlockTests
#endif
#include <boost/test/unit_test.hpp>

#include "../haplo/subblock_cpu.hpp"
#include "../haplo/tree_gpu.h"
#include "hip/hip_runtime.h"

using namespace std::chrono;

static constexpr const char* input_100_10  = "new_outputs/geraci/100_10_0.1_0.4/output_5298.txt";

BOOST_AUTO_TEST_SUITE( TreeGpuSuite )

BOOST_AUTO_TEST_CASE( canCreateTree )
{
    using block_type    = haplo::Block<5298, 4, 4>;
    using subblock_type = haplo::SubBlock<block_type, 4, 4, haplo::devices::cpu>;
    using tree_type     = haplo::Tree<subblock_type, haplo::devices::gpu>;
    
    block_type      block(input_100_10);
    subblock_type   sub_block(block, 1);
    sub_block.print();

    // For now just use the first device 
    // Later use device manager
    size_t device_index = 0;
    
    tree_type tree(sub_block, device_index);
    
    // Search the tree for the haplotype 
    tree.search();
}

BOOST_AUTO_TEST_SUITE_END()
