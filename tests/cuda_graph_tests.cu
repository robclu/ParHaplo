// ----------------------------------------------------------------------------------------------------------
/// @file   cuda_graph_tests.cu
/// @brief  Test suite for parahaplo GPU graph search tests
// ----------------------------------------------------------------------------------------------------------

#define BOOST_TEST_DYN_LINK
#ifdef STAND_ALONE
    #define BOOST_TEST_MODULE CudaGraphTests
#endif
#include <boost/test/unit_test.hpp>

#include "../haplo/subblock_cpu.hpp"
#include "../haplo/graph_gpu.h"
#include "hip/hip_runtime.h"

using namespace std::chrono;

static constexpr const char* test_input   = "output_files/output_simulated_265.txt";
static constexpr const char* input_zero   = "input_files/input_zero.txt";

BOOST_AUTO_TEST_SUITE( GraphGpuSuite )

BOOST_AUTO_TEST_CASE( canCreateGraph )
{
    using block_type    = haplo::Block<265, 4, 4>;
    using subblock_type = haplo::SubBlock<block_type, 4, 4, haplo::devices::cpu>;
    using graph_type    = haplo::Graph<subblock_type, haplo::devices::gpu>;
   
    block_type block(test_input);
    
    std::cout << "NUM_SUB_BLOCKS " <<  block.num_subblocks() << "\n";

    subblock_type sub_block(block, 1);
    sub_block.print();

    // For now just use the first device 
    // Later use device manager
    size_t device_index = 0;
    
    graph_type graph(sub_block, device_index);
    
    // Search the tree for the haplotype 
    graph.search();
}

BOOST_AUTO_TEST_SUITE_END()
