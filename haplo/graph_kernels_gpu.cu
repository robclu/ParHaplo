#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "data.h"
#include "debug.h"
#include "graph_internal.h"
#include "math.h"

#define BLOCK_SIZE 1024
//#undef DEBUG

namespace haplo {

using graph_type = internal::Graph;
using data_type  = Data;

extern __shared__ size_t distance[];
extern __shared__ uint8_t shared_sets[];

__device__
void map_distances(data_type& data, graph_type& graph, const size_t threads)
{
    const size_t total_elems  = data.reads * (data.reads - 1) / 2;
    const size_t read_idx_one = blockIdx.x / (data.reads - 1);
    const size_t subt_elems   = (data.reads - read_idx_one) * (data.reads - read_idx_one - 1) / 2;
    const size_t read_idx_two = read_idx_one + (blockIdx.x - (total_elems - subt_elems)) + 1;
    const size_t snp_idx      = threadIdx.y;

    const auto read_info_one = data.read_info[read_idx_one];    
    const auto read_info_two = data.read_info[read_idx_two];    
    
    bool    first_valid = false , second_valid = false;
    uint8_t first_value = 0     , second_value = 0    ;

    if (read_idx_one < data.reads && read_idx_two < data.reads) {    
        // Check if each of the values is valid
        if (read_info_one.start_index() <= snp_idx && read_info_one.end_index() >= snp_idx) {
            first_valid = true;
            first_value = data.data[read_info_one.offset() + snp_idx - read_info_one.start_index()];
        }
        if (read_info_two.start_index() <= snp_idx && read_info_two.end_index() >= snp_idx) {
            second_valid = true;
            second_value = data.data[read_info_two.offset() + snp_idx - read_info_two.start_index()];
        }
        
        // Set the values for the graph edge
        graph.edges[blockIdx.x].f1 = read_idx_one;
        graph.edges[blockIdx.x].f2 = read_idx_two;
    } 
    
    // Load the distance into the shared array
    if (first_valid && second_valid) {
        if (first_value != second_value && first_value <= 1 && second_value <= 1) {
            distance[snp_idx] = 10; distance[snp_idx + threads] = 1;
        } else if (first_value != second_value && (first_value <= 1 || second_value <= 1)) {
            distance[snp_idx] = 5; distance[snp_idx + threads] = 1;
        } else if (first_value == second_value && first_value <= 1 && second_value <= 1) {
            distance[snp_idx] = 0; distance[snp_idx + threads] = 1;
        } else if (first_value == second_value & first_value > 1 && second_value > 1) {
            distance[snp_idx] = 0; distance[snp_idx + threads] = 0;
        }   
    } else if (first_valid && !second_valid && first_value <= 1) {
        distance[snp_idx] = 5; distance[snp_idx + threads] = 1;
    } else if (second_valid && !first_valid && second_value <= 1) {
        distance[snp_idx] = 5; distance[snp_idx + threads] = 1;
    } else if (!first_valid && !second_valid) {
        distance[snp_idx] = 0; distance[snp_idx + threads] = 0;
    }
    __syncthreads();    
}

__device__
void reduce_distances(data_type& data, graph_type& graph, const size_t threads) 
{
    size_t reduction_threads = threads;
    
    while (reduction_threads > 1) {
        size_t other_idx_one = threadIdx.y + (reduction_threads / 2);
        size_t other_idx_two = other_idx_one + threads;
        
        if (threadIdx.y < (reduction_threads / 2)) {                
            distance[threadIdx.y] += distance[other_idx_one];
            distance[threadIdx.y + threads] += distance[other_idx_two];
        }
        if (reduction_threads % 2 == 1) {
            // Load the extra value
            if (threadIdx.y == reduction_threads / 2) {
                distance[threadIdx.y] = distance[threadIdx.y + reduction_threads / 2];
                distance[threadIdx.y + threads] = distance[threadIdx.y + threads + reduction_threads / 2];
            }
            reduction_threads /= 2; reduction_threads += 1;
        } else reduction_threads /= 2;
        __syncthreads();
    }
    // Set the weight of the edge
    if (distance[threads] > 0 && threadIdx.y == 0) {
        graph.edges[blockIdx.x].distance = static_cast<float>(distance[0] / 10.f) /
                                           static_cast<float>(distance[threads])  + 0.5f;
    } else if (threadIdx.y == 0 && distance[threads] == 0) {
        graph.edges[blockIdx.x].distance = 1.0f;
    }
    
    // Modify the value so that it's easier to sort 
    if (graph.edges[blockIdx.x].distance == 1.0f && threadIdx.y == 0) {
        graph.edges[blockIdx.x].distance = 0.0f;
    }
}
 
__device__
void swap_edges(graph_type& graph, const size_t edge_idx_one, const size_t edge_idx_two)
{
    Edge temp = graph.edges[edge_idx_one];
    graph.edges[edge_idx_one] = graph.edges[edge_idx_two];
    graph.edges[edge_idx_two] = temp; 
}

// Out to in operation for bitonic sort
__global__ 
void bitonic_out_in_sort(graph_type graph, const size_t block_size, const size_t total_elements)
{
    const size_t block_idx  = blockIdx.x / (block_size / 2);
    const size_t idx_one    = blockIdx.x + (block_idx * (block_size / 2));
    const size_t idx_two    = idx_one + (block_size - (blockIdx.x % (block_size / 2)) - 1) 
                            - (idx_one % (block_size / 2));
    
    // If the dimensions are in range
    if (idx_one < total_elements && idx_two < total_elements && threadIdx.y == 0) {
        // The edges need to be swapped if the right one is larger than the left one
        // or if the left one has a value of 0.5, since those must be removed
        if (graph.edges[idx_one].distance <= graph.edges[idx_two].distance) {
            swap_edges(graph, idx_one, idx_two);
        } 
    }
}

// Out to out operation fr bitonic sort
__global__
void bitonic_out_out_sort(graph_type graph, const size_t block_size, const size_t total_elements)
{
    const size_t block_idx = blockIdx.x / (block_size / 2);
    const size_t idx_one   = blockIdx.x + (block_idx * (block_size / 2));
    const size_t idx_two   = idx_one + (block_size / 2);

    // Check that the node index is in the first half of the block and the comp node is in range
    if (idx_one < total_elements && idx_two < total_elements && threadIdx.y == 0) {
        // The edges need to be swapped if the right one is larger than the left one
        // or if the left one has a value of 0.5, since those must be removed
        if (graph.edges[idx_one].distance <= graph.edges[idx_two].distance) {
            swap_edges(graph, idx_one, idx_two);
        }
    }    
}

__device__ 
size_t find_last_valid_edge(const graph_type& graph)
{
    bool   found_end       = false;   
    size_t last_valid_edge = 0;
    while (!found_end) {
       if (graph.edges[last_valid_edge].distance != 0.0f) 
            ++last_valid_edge;
        else 
            found_end = true;
    }
    return last_valid_edge;
}

// Prints the edges of a node
__global__
void print_edges(data_type data, graph_type graph)
{
    if (threadIdx.y == 0 && blockIdx.x == 0) {   
        for (size_t i = 0; i < data.reads * (data.reads -1) / 2; ++i) {
            if (graph.edges[i].distance != 0.0f) {
                printf("%.4f ", graph.edges[i].distance);
                printf("%i ", graph.edges[i].f1);
                printf("%i\n", graph.edges[i].f2);
            }
        }
    } 
}

__global__
void search_graph(data_type data, graph_type graph, size_t threads) 
{
    const size_t total_vertices = data.reads * (data.reads - 1) / 2;
    
    map_distances(data, graph, threads);
    reduce_distances(data, graph, threads);
}

// For now we assume the number of partitions is less than 6000
// Add dynamic parallelism
template <uint8_t Set> __device__ 
uint8_t in_set(graph_type&  graph        , const uint32_t fragment, const size_t block_idx_start, 
               const size_t block_idx_end, uint8_t*      shared_set)
{ 
    // Need to extend this to use multiple blocks 
    const size_t frag_idx           = threadIdx.y;
    size_t       reduction_threads  = Set == 1 ? graph.set_one_size : graph.set_two_size;

    if (blockIdx.x == 0 && threadIdx.y == 0) {
        for (size_t i = 0; i < reduction_threads; ++i) {
            if (Set == 1 ? graph.set_one[i] == fragment : graph.set_two[i] == fragment) {
                return true;
            }
        }
        return false;
    }
/*
    if (blockIdx.x >= block_idx_start && blockIdx.x < block_idx_end && frag_idx < reduction_threads) {    
        shared_set[frag_idx] = Set == 1 ? graph.set_one[frag_idx] == fragment 
                                        : graph.set_two[frag_idx] == fragment;
        __syncthreads();
        
        // Perform the reduction  
        while (reduction_threads > 1) {
            if (frag_idx < (reduction_threads / 2)) {
                // If the thread is part of the first half of the threads
                shared_set[frag_idx] += shared_set[frag_idx + (reduction_threads / 2)];
            } 
            if (reduction_threads % 2 == 1) {
                if (frag_idx == reduction_threads / 2) {
                    // If there are an odd number of elements then move the last element
                    shared_set[frag_idx] = shared_set[frag_idx + (reduction_threads / 2)];
                }
                reduction_threads /= 2; reduction_threads += 1;
            } else reduction_threads /= 2;               
            __syncthreads();
        }
    }
    if (threadIdx.y == 0 && blockIdx.x == 0) result = shared_set[0];
    if (frag_idx < Set == 1 ? graph.set_one_size : graph.set_two_size) shared_set[frag_idx] = 0;
    __syncthreads();
    
    return result;
*/
}



__device__
void partition_next_largest_fragment(graph_type& graph, size_t& last_set_edge, const size_t last_valid_edge,
                                     uint8_t* shared_set)
{
    const size_t initial_edge = last_set_edge;
    bool   found              = false;
   
    if (blockIdx.x == 0 && threadIdx.y == 0) { 
        while (last_set_edge < last_valid_edge - 1 && !found) {
            uint8_t f1_in_set_1 = in_set<1>(graph, graph.edges[last_set_edge].f1, 0, 1, shared_set); 
            uint8_t f1_in_set_2 = in_set<2>(graph, graph.edges[last_set_edge].f1, 0, 1, shared_set); 
            uint8_t f2_in_set_1 = in_set<1>(graph, graph.edges[last_set_edge].f2, 0, 1, shared_set); 
            uint8_t f2_in_set_2 = in_set<2>(graph, graph.edges[last_set_edge].f2, 0, 1, shared_set);  
            
            if (f1_in_set_1 && !f2_in_set_2 && !f2_in_set_1) {
                // f1 in set one, add f2 to set two
                graph.set_two[graph.set_two_size] = graph.edges[last_set_edge].f2;
                ++graph.set_two_size;
                found = true;
            } else if (f2_in_set_1 && !f1_in_set_2 && !f1_in_set_1) {
                // f2 in set one, add f1 to set two
                graph.set_two[graph.set_two_size] = graph.edges[last_set_edge].f1;
                ++graph.set_two_size;
                found = true;
            } else if (f1_in_set_2 && !f2_in_set_1 && !f2_in_set_2) {
                // f1 in set 2, add f2 so set one
                graph.set_one[graph.set_one_size] = graph.edges[last_set_edge].f2;
                ++graph.set_one_size;
                found = true;
            } else if (f2_in_set_2 && !f1_in_set_1 && !f1_in_set_2) {
                // f2 in set 2, add f1 to set one
                graph.set_one[graph.set_one_size] = graph.edges[last_set_edge].f1;
                ++graph.set_one_size;
                found = true;
            } else ++last_set_edge;
        }
        if (found) {
            Edge temp                   = graph.edges[initial_edge];
            graph.edges[initial_edge]   = graph.edges[last_set_edge];
            graph.edges[last_set_edge]  = temp;
            last_set_edge               = initial_edge + 1;
        } else last_set_edge = initial_edge;
    }
}

__device__
void partition_next_smallest_fragment(graph_type& graph, size_t& last_set_edge, uint8_t* shared_set)
{
    const size_t initial_edge = last_set_edge;
    bool   found = false;
    
    if (blockIdx.x == 0 && threadIdx.y == 0) { 
        while (last_set_edge >= 1 && !found) {
            uint8_t f1_in_set_1 = in_set<1>(graph, graph.edges[last_set_edge].f1, 0, 1, shared_set); 
            uint8_t f1_in_set_2 = in_set<2>(graph, graph.edges[last_set_edge].f1, 0, 1, shared_set); 
            uint8_t f2_in_set_1 = in_set<1>(graph, graph.edges[last_set_edge].f2, 0, 1, shared_set); 
            uint8_t f2_in_set_2 = in_set<2>(graph, graph.edges[last_set_edge].f2, 0, 1, shared_set); 
            
            if (f1_in_set_1 && !f2_in_set_1 && !f2_in_set_2) {
                // f1 in set one, add f2 to set one
                graph.set_one[graph.set_one_size] = graph.edges[last_set_edge].f2;
                ++graph.set_one_size;
                found = true;
            } else if (f2_in_set_1 && !f1_in_set_1 && !f1_in_set_2) {
                // f2 in set one, add f1 to set one
                graph.set_one[graph.set_one_size] = graph.edges[last_set_edge].f1;
                ++graph.set_one_size;
                found = true;
            } else if (f1_in_set_2 && !f2_in_set_2 && !f2_in_set_1) {
                // f1 in set 2, add f2 so set two
                graph.set_two[graph.set_two_size] = graph.edges[last_set_edge].f2;
                ++graph.set_two_size;
                found = true;
            } else if (f2_in_set_2 && !f1_in_set_2 && !f1_in_set_1) {
                // f2 in set 2, add f1 to set one
                graph.set_two[graph.set_two_size] = graph.edges[last_set_edge].f1;
                ++graph.set_two_size;
                found = true;
            } else --last_set_edge;
        }
        if (found) {
            Edge temp                   = graph.edges[initial_edge];
            graph.edges[initial_edge]   = graph.edges[last_set_edge];
            graph.edges[last_set_edge]  = temp;
            last_set_edge               = initial_edge - 1; 
        } else last_set_edge = initial_edge;
    }
}

__device__ 
void print_sets(graph_type& graph)
{
    if (threadIdx.y == 0 && blockIdx.x == 0) {
        for (size_t i = 0; i < graph.set_one_size; ++i) 
            printf("%i ", graph.set_one[i]);
        printf("\n");
        for (size_t i = 0; i < graph.set_two_size; ++i) 
            printf("%i ", graph.set_two[i]);
        printf("\n"); 
    }
}

__global__ 
void map_to_partitions(data_type data, graph_type graph)
{
    const size_t last_valid_edge        = find_last_valid_edge(graph);
    size_t       last_set_edge_forward  = 1;
    size_t       last_set_edge_backward = last_valid_edge - 1;
    bool         keep_partitioning      = true;
   
    extern __shared__ uint8_t sets[];
    
#ifdef DEBUG
    if (threadIdx.y == 0 && blockIdx.x == 0)
        printf("LVE : %i\n", last_valid_edge);
#endif
    
    // Add the first elements in the partitions
    graph.set_one[0] = graph.edges[0].f1 + 1; graph.set_one_size = 1;
    graph.set_two[0] = graph.edges[0].f2 + 1; graph.set_two_size = 1;
    
    // Partition the remaining fragments
    if (threadIdx.y == 0 && blockIdx.x == 0) {
        while (keep_partitioning) {
            size_t last_edge_back_before = last_set_edge_backward;
            size_t last_edge_fwd_before  = last_set_edge_forward;

            partition_next_largest_fragment(graph, last_set_edge_forward, last_valid_edge, &sets[0]);
            partition_next_smallest_fragment(graph, last_set_edge_backward, &sets[0]);   
            
            if (last_set_edge_forward == last_valid_edge || last_set_edge_backward == 1                           ||
                (last_set_edge_forward == last_edge_fwd_before && last_set_edge_backward == last_edge_back_before) ||
                (graph.set_one_size + graph.set_two_size == data.reads)                                           ) {
                keep_partitioning = false;
            }
        }  
    }
    
    print_sets(graph);
}

}               // End namespace haplo
