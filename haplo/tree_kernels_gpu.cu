#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include "tree_internal.h"

namespace haplo {

struct BoundsGpu {

    size_t lower;               // The lower value for the bound    
    size_t upper;               // The upper value for the bound
    size_t diff;                // The difference between the upper and lower bound
    size_t index;               // The snp index the bound represents
    
    // Overlaod the equality operator
    __device__
    void operator=(BoundsGpu& other) 
    {
        lower           = other.lower;
        upper           = other.upper;
        diff            = other.diff;
        index           = other.index;
    }
};

// Maps all the unsearched snps to a n array of BoundsGpu structs which can then be reduces
__global__ 
void map_unsearched_snps(internal::Tree tree, BoundsGpu* snp_bounds, const size_t comp_node_idx)
{
    size_t read_start = 0, read_end = 0, row_offset = 0, opp = 0, same = 0;
    
    const size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Get a reference to the bound parameter
    BoundsGpu* bounds = &snp_bounds[thread_id];
    
    // Index of the haplotype for the comparison
    const size_t ref_haplo_idx = tree.search_snps[thread_id + tree.last_searched_snp + 1];
    bounds->index              = ref_haplo_idx;
            
    // Get a pointer to the node
    const TreeNode* comp_node = &tree.nodes[comp_node_idx];
    
    // Go back up the tree and determine the 
    for (size_t i = tree.last_searched_snp + 1; i > 0; --i) {
         
        // DEBUG  
        printf("Node HI : %i ,", comp_node->haplo_idx);
        printf("Ref HI : %i\n" , ref_haplo_idx);
        
        // Go through the alignments for the node
        for (size_t j = 0; j < comp_node->alignments; ++j) {
            read_start = tree.read_info[comp_node->read_ids[j]].start_index();
            read_end   = tree.read_info[comp_node->read_ids[j]].end_index();
            
            // If any of the comp and ref snps (nodes) have overlapping positions
            if (read_start <= comp_node->haplo_idx && read_start <= ref_haplo_idx &&
                read_end   >= comp_node->haplo_idx && read_end   >= ref_haplo_idx ) {
                row_offset = tree.read_info[comp_node->read_ids[j]].offset();

                uint8_t comp_value = tree.data[row_offset + comp_node->haplo_idx];
                uint8_t ref_value  = tree.data[row_offset + ref_haplo_idx];
                
                if (comp_value == ref_value && ref_value <= 1) {
                    ++same;
                    // DEBUG
        //            printf("Adding Same : %i, %i\n", comp_node->read_ids[j], ref_haplo_idx);
                } else if (comp_value != ref_value && ref_value <= 1 && comp_value <=1) {
                    ++opp;
                    // DEBUG
        //            printf("Adding Oppp : %i, %i\n", comp_node->read_ids[j], ref_haplo_idx);
                }
            }
        }
        if (i > 1) {
            // Move the pointer up the tree (towards the root)
            const size_t root_idx = comp_node->root_idx;
            comp_node = tree.node_ptr(root_idx);
            
            // DEBUG
          //  printf("Reassign : Root ID : %i : Node ID :\n", root_idx, comp_node->node_idx); 
        }
    }
    // Format the bounds 
    bounds->lower = min((unsigned int)same, (unsigned int)opp);
    bounds->upper = max((unsigned int)same, (unsigned int)opp);
    bounds->diff  = bounds->upper - bounds->lower;
    
    // DEBUG
    //printf("Bounds Diff Inline: %i\n", bounds->diff);
}

// Very similar to the above function, but does the mapping for each left node of a level
__device__ 
void map_leaf_bounds(internal::Tree& tree, const size_t node_idx)
{
    size_t          elements_used = 0, row_offset = 0;
    uint8_t         ref_value     = 0;
    TreeNode* const ref_node      = tree.node_ptr(node_idx);
    const TreeNode* comp_node     = &tree.nodes[ref_node->root_idx];
    
    for (size_t i = tree.last_searched_snp; i > 0; --i) {
        for (size_t j = 0; j < comp_node->alignments; ++j) {
            row_offset = tree.read_info[comp_node->read_ids[j]].offset();
            ref_value  = tree.data[row_offset + ref_node->haplo_idx];
             
            if ((ref_value == ref_node->value && ref_value <= 1 && comp_node->read_ids[j] == 1) ||
                (ref_value != ref_node->value && ref_value <= 1 && comp_node->read_ids[j] == 0)) {
                    // Optimal selection -- reduce the upper bound, dont increase lower bound
                    --ref_node->ubound; ++elements_used;
            } else if ((ref_value != ref_node->value && ref_value <= 1 && comp_node->read_ids[j] == 1) ||
                       (ref_value == ref_node->value && ref_value <= 1 && comp_node->read_ids[j] == 0)) {
                    // Non-optimal selection -- incread lower bound, don't reduce upper bound
                    ++ref_node->lbound; ++elements_used;
            }
        }
        if (i > 1) {
            comp_node = tree.node_ptr(comp_node->root_idx);
            
            // DEBUG 
            printf("Moving Up Tree\n");
        }
        
    }
    // For all remaining elements, we can reduce the upper bound
    ref_node->ubound -= (tree.snp_info[ref_node->haplo_idx].elements() - elements_used);
}

// Checks which of the two snps is more "vaulable", first by the bounds diff
// parameters, and then by the snp index
__device__
bool more_valuable(BoundsGpu* snp_one, BoundsGpu* snp_two)
{
    // First check which has the greater differenece between upper and lower
    return snp_one->diff > snp_two->diff 
                         ? true 
                         : snp_two->diff > snp_one->diff 
                            ? false 
                            : snp_one->index < snp_two->index 
                                ? true : false;
}

// "Reduce" function for the list of selection params to determine the best param 
//  The reduction is done such that the resulting bound has the highest diff and 
//  lowest index (this is to mean that the snp is most correlated to the snps already
//  searched and is also closest to them (hence the lowest index))
__global__ 
void reduce_unsearched_snps(BoundsGpu* snp_bounds, const size_t elements)
{
    const size_t thread_id      = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t reductions     = (size_t)(ceil(log2((double)elements)));
    size_t reduction_threads    = elements - 1;
    size_t snp_idx_other        = 0;
    
    while (reduction_threads > 1) {
        // Only the first half of the threads do the reduction 
        snp_idx_other = thread_id + (reduction_threads / 2);
        
        if (thread_id < (reduction_threads / 2)) {
            // If the more rightward bound is more valuable 
            if (!more_valuable(&snp_bounds[thread_id], &snp_bounds[snp_idx_other])) {
                // We need ro replace the left value withe the right one
                BoundsGpu temp            = snp_bounds[thread_id];
                snp_bounds[thread_id]     = snp_bounds[snp_idx_other];
                snp_bounds[snp_idx_other] = temp;
            }
        }
        // If we came from an odd number of bounds, the last one just fetches a value
        if (reduction_threads % 2 == 1) {
            if (thread_id == (reduction_threads / 2)) {
                // There is an odd number of elements in the array,
                // The last thread just needs to move a value 
                BoundsGpu temp          = snp_bounds[thread_id];
                snp_bounds[thread_id]  = snp_bounds[snp_idx_other];
                snp_bounds[snp_idx_other] = temp;
            }
            reduction_threads /= 2; reduction_threads += 1;
        } else reduction_threads /= 2;
        __syncthreads();
    }
}


// Add any unaligned reads to a node (snp site)
__device__ 
void add_alignments(internal::Tree& tree, const size_t node_idx, const size_t thread_id)
{
    size_t read_offset = 0, align_count = 0, element_position = 0;
    
    // Get a pointer to the node
    TreeNode* node = tree.node_ptr(node_idx);
   
    // DEBUG
    //printf("AlignS : %i\n", node->haplo_idx);
    
    // Allocate some temp space for the node alignments (this is usually small)
    size_t* alignments = new size_t[tree.snp_info[node->haplo_idx].elements()]; 
    size_t* values     = new size_t[tree.snp_info[node->haplo_idx].elements()];
    size_t* indices;
    
    if (thread_id == 0) {
        indices = new size_t[tree.snp_info[node->haplo_idx].elements()];
    }
                
    // Go through all the unaligned reads
    for (size_t i = tree.last_unaligned_idx; i < tree.reads; ++i) {
        if (tree.aligned_reads[i] >= tree.snp_info[node->haplo_idx].start_index() &&
            tree.aligned_reads[i] <= tree.snp_info[node->haplo_idx].end_index()  ) {
        
            // If the row crosses the snp
            if (tree.read_info[tree.aligned_reads[i]].start_index() <= node->haplo_idx &&
                tree.read_info[tree.aligned_reads[i]].end_index()   >= node->haplo_idx ) {
            
                // Get the offset in memory of the start of the read
                read_offset         = tree.read_info[tree.aligned_reads[i]].offset();
                element_position    = node->haplo_idx - tree.read_info[tree.aligned_reads[i]].start_index();
                auto element = tree.data[read_offset + element_position];
            
                // Do the alignement for the reads to this node
                if ((element == 0 && node->value == 0) || (element == 1 && node->value == 1)) {
                    // DEBUG
     //               printf("Align1 : %i\n", i);
                    values[align_count] = 1; alignments[align_count++] = tree.aligned_reads[i];
                    if (thread_id == 0) indices[align_count - 1] = i; 
                } else if ((element == 0 && node->value == 1) || (element == 1 && node->value == 0)) {
                    values[align_count] = 0; alignments[align_count++] = tree.aligned_reads[i];
                    if (thread_id == 0) indices[align_count - 1] = i; 
                    // DEBUG
     //               printf("Align2 : %i\n", i);
                }
            }
        }
    }
   
    // Move the found alignments to the node
    node->alignments     = align_count;
    node->read_ids       = new size_t[align_count];
    node->read_values    = new uint8_t[align_count];
   
    if (thread_id == 0) node->indices = new size_t[align_count];
    
    for (size_t i = 0; i < align_count; ++i) {
        node->read_ids[i]    = alignments[i];
        node->read_values[i] = (uint8_t)values[i];
        if (thread_id == 0) node->indices[i] = indices[i];
    }
    // Clean memory
    free(alignments); free(values);
    if (thread_id == 0) free(indices);
}


__global__ 
void map_level(internal::Tree tree            , BoundsGpu* snp_bounds, const size_t prev_level_start, 
               const size_t   this_level_start)
{
    // Set node parameters
    size_t      node_idx  = threadIdx.x + this_level_start;
    TreeNode*   node      = tree.node_ptr(node_idx);
    TreeNode*   root_node = tree.node_ptr(prev_level_start + (threadIdx.x / 2));
   
    printf("Reached!\n");
    
    // Set some of the node parameters
    node->haplo_idx = snp_bounds[0].index;
    node->root_idx  = prev_level_start + (threadIdx.x / 2);
    node->lbound    = root_node->lbound;
    node->ubound    = root_node->ubound;
    node->value     = threadIdx.x % 2 == 0 ? 0 : 1;
  
    // DEBUG
   // printf("H ID : %i\n", node->haplo_idx);
    
    // DEBUG
    printf("Lbound : %i " , node->lbound); 
    printf("Ubound : %i\n", node->ubound); 
    
    // Update the bounds for the node 
    map_leaf_bounds(tree, node_idx);
  
    // DEBUG
    printf("Lbound : %i " , node->lbound); 
    printf("Ubound : %i\n", node->ubound); 

    // Add the alignments for the node 
    add_alignments(tree, node_idx, threadIdx.x);
    
    // DEBUG
    printf("Alignments : %i\n", node->alignments);
}

__device__ 
void swap_search_snp_indices(internal::Tree& tree, const size_t swap_idx)
{
    ++tree.last_searched_snp;
    if (swap_idx != tree.last_searched_snp) {
        const size_t temp                        = tree.search_snps[tree.last_searched_snp];
        tree.search_snps[tree.last_searched_snp] = tree.search_snps[swap_idx];
        tree.search_snps[swap_idx]               = temp;
    }
}


// Updates the array of reads which have been aligned
__device__
void update_global_alignments(internal::Tree& tree, const size_t node_idx) 
{
    // DEBUG 
    printf("Node ID Align : %i\n", node_idx);
    
    const TreeNode* const node = tree.node_ptr(node_idx);
   
    printf("Aligns: %i\n", node->alignments);
    
    for (size_t i = tree.last_unaligned_idx; i < tree.last_unaligned_idx + node->alignments; ++i) {
        size_t temp = tree.aligned_reads[node->indices[i - tree.last_unaligned_idx]];
        tree.aligned_reads[node->indices[i - tree.last_unaligned_idx]] = tree.aligned_reads[i];
        tree.aligned_reads[i] = temp;
    } 
    tree.last_unaligned_idx += node->alignments;
    printf("Finished Align!\n");
}


__device__ size_t       prev_level_start = 0;           // The index of the first nod in the previous level
__device__ size_t       this_level_start = 1;           // For each level, this is the index in the 
                                                        // node array of the first element in the level 
__device__ size_t       nodes_in_level   = 2;           // The number of nodes (sub-branches) in the level
__device__ size_t       unsearched_snps;                // The number of unsearched snps
__device__ size_t       comp_node_idx;                  // The index of the comparison node for node seletion

__global__ void search_tree(internal::Tree tree, BoundsGpu* snp_bounds, size_t start_ubound, size_t device_index)
{
    // DEBUG 
    printf("Device Index : %i\n", device_index);
    printf("Start Bound  : %i\n" , start_ubound);
    
    struct hipDeviceProp_t device_properties;                // So that we can know the max number of threads

    // ---------------------------------------- ROOT NODE -------------------------------------------------

    // DEBUG
    for (size_t i = 0; i < tree.reads; ++i) printf("%i\n", tree.aligned_reads[i]);
    for (size_t i = 0; i < tree.snps;  ++i) printf("%i\n", tree.search_snps[i]);
    
    TreeNode& node = tree.nodes[0];
    node.haplo_idx = tree.search_snps[tree.last_searched_snp];
    node.node_idx  = 0; node.value  = 0;
    
    // Set the alignments for the tree root
    add_alignments(tree, 0, 0);

    // Add the alignments to the overall alignments
    update_global_alignments(tree, 0);
    
    // DEBUG
    for (size_t i = 0; i < tree.last_unaligned_idx; ++i) {
        printf("%i ", tree.aligned_reads[i]);
    } printf("\n");

    // Go over all the nodes that have not been searched and see how correlated they are
    comp_node_idx   = node.node_idx; 
    unsearched_snps = tree.snps - tree.last_searched_snp - 1;

    // Perform a "mapping" step to map all the unsearched snps to their potenetial bounds
    map_unsearched_snps<<<1, unsearched_snps>>>(tree, snp_bounds, comp_node_idx);
    if (hipSuccess != hipGetLastError()) printf("Kernel Launch Error for SNP Map!\n");
    hipDeviceSynchronize(); 
    __syncthreads();
    
    // Do the reduction to get the index of the next node to search
    reduce_unsearched_snps<<<1, unsearched_snps>>>(snp_bounds, unsearched_snps);
    hipDeviceSynchronize();
    __syncthreads();
    
    --unsearched_snps; // We have searched another snp
    
    // DEBUG 
    for (size_t i = 0; i < unsearched_snps; ++i) {
        printf("%i ", snp_bounds[i].diff);
    } printf("\n");
    for (size_t i = 0; i < unsearched_snps; ++i) {
        printf("%i ", snp_bounds[i].index);
    } printf("\n");
    
    // Reduction has found the next node, modify the searned snp array
    swap_search_snp_indices(tree, snp_bounds[0].index);
    
    // Update the bounds for the tree
    node.lbound = 0; node.ubound = start_ubound - tree.snp_info[node.haplo_idx].elements();
    
    // ----------------------------------- STARTING TREE SEARCH -------------------------------------------------
   
    // Iterate through level
    size_t iteration = 0;
    while (tree.last_searched_snp < tree.snps && counter++ < 13) {
        if (hipSuccess != hipGetLastError()) printf("Error!\n");
       
        printf("Starting new iteration!\n");
        
        // We can only launch 
        // Perform a "mapping" step, which maps the bounds onto the nodes in the level
        map_level<<<1, nodes_in_level>>>(tree, snp_bounds, prev_level_start, this_level_start);
        if (hipSuccess != hipGetLastError()) printf("Kernel Launch Error for Level Map!\n");
        hipDeviceSynchronize();
        __syncthreads();
        
        // Add the new alignments to the array of overall aligned reads -- use first node
        if (threadIdx.x == 0) {
            update_global_alignments(tree, this_level_start);
        }
        __syncthreads(); 
        
        // Now "reduce" the level, which is essentially a pruning step
        
        // Map unsearched snps
        map_unsearched_snps<<<1, unsearched_snps>>>(tree, snp_bounds, this_level_start);
        if (hipSuccess != hipGetLastError()) printf("Kernel Launch Error for SNP Map!\n");
        hipDeviceSynchronize();
        __syncthreads();
       
        // DEBUG
        printf("Before\n");
        for (size_t i = 0; i < unsearched_snps; ++i) {
            printf("%i ", snp_bounds[i].diff);
        } printf("\n");
        for (size_t i = 0; i < unsearched_snps; ++i) {
            printf("%i ", snp_bounds[i].index);
        } printf("\n");
            
        reduce_unsearched_snps<<<1, unsearched_snps>>>(snp_bounds, unsearched_snps);
        if (hipSuccess != hipGetLastError()) printf("Kernel Launch Error for SNP Reduce!\n");
        hipDeviceSynchronize();
     
        if (threadIdx.x == 0) {   
            --unsearched_snps;
        }
        __syncthreads();
        
        // DEBUG
        for (size_t i = 0; i < unsearched_snps; ++i) {
            printf("%i ", snp_bounds[i].diff);
        } printf("\n");
        for (size_t i = 0; i < unsearched_snps; ++i) {
            printf("%i ", snp_bounds[i].index);
        } printf("\n");

        // DEBUG
        printf("Aligned Reads : %i\n", tree.last_unaligned_idx);
        for (size_t i = 0; i < tree.reads; ++i) {
            printf("%i ", tree.aligned_reads[i]);
        } printf("\n");
       
        printf("Lowest Index :%i\n", snp_bounds[0].index);
        
        // Swap the indices of the last unsearched and the about to be searched snps
        swap_search_snp_indices(tree, snp_bounds[0].index);
        
        // DEBUG
        printf("Searched SNPS :\n");
        for (size_t i = 0; i < tree.snps; ++i) {
            printf("%i ", tree.search_snps[i]);
        } printf("\n");
        
        // Change the indices of the start and end of the level
        prev_level_start  = this_level_start;
        this_level_start += nodes_in_level;
        nodes_in_level   *= 2;
        
        printf("Nodes in level : %i\n", nodes_in_level);
        printf("Start Node     : %i\n", this_level_start);
        printf("Last Node      : %i\n", this_level_start + nodes_in_level);
        printf("Last SNP       : %i\n", tree.last_searched_snp);
    }
        
        
    /*
    size_t max = 0, index = 0; BoundsGpu bounds_temp, bounds_final;
    for (size_t i = tree.last_searched_snp + 1; i < tree.snps; ++i) {yy
        bounds_temp = compare_snps(tree, node.node_idx, i);
        if (bounds_temp.diff > max) { 
            index        = i; 
            max          = bounds_temp.diff; 
            bounds_final = bounds_temp;
        }
        printf("Result : %i\n", bounds_temp.diff);
        printf("Max    : %i\n", max   );
        printf("Index  : %i\n", index );
   }
  
    // DEBUG
    printf("Most Correlated : %i ", index);
    printf("Correlation : %i\n"   , max  );
    
    // The first node has now been searched
    tree.last_searched_snp++;
    
    // Swap the SNPs here
   
    // Set the upper and lower bounds based on the most correlated node
    node.lbound += bounds_final.lower; node.ubound -= bounds_final.upper;
    
   */
/*    
    // Make the next 2 nodes point back to this one
    TreeNode& left_child = tree.nodes[1]; TreeNode& right_child = tree.nodes[2];
    left_child.root_idx  = 0; right_child.root_idx  = 0;  
    left_child.value     = 1; right_child.value     = 1;
    left_child.node_idx  = 1; right_child.node_idx  = 2;
        
    printf("%i rest\n ",*result);
    
    left_child.haplo_idx = 1; right_child.haplo_idx  = 1;
    printf("%i rest\n ", left_child.node_idx);
    
    add_alignments(tree, tree.read_info, tree.reads, tree.snp_info, tree.data, &left_child , aligned, tree.last_aligned);
    add_alignments(tree, tree.read_info, tree.reads, tree.snp_info, tree.data, &right_child, aligned, tree.last_aligned);
   
    printf("%i rest\n ", left_child.node_idx);

    //left_child.lbound    = bounds.same; left_child.ubound  = *ubound - (bounds.same + left_child.alignments);
    //right_child.lbound   = bounds.opps; right_child.ubound = *ubound - (bounds.opps + right_child.alignments);
    
    // Add the alignments due to these nodes -- same if we use right or left child
    for (size_t i = tree.last_aligned; i < tree.last_aligned + left_child.alignments; ++i) {
            aligned[left_child.read_ids[i - tree.last_aligned]] = aligned[i];
            aligned[i] = left_child.read_ids[i - tree.last_aligned];
    } tree.last_aligned += left_child.alignments;

    printf("%i rest\n ", left_child.node_idx);

    temp = 0;
    for (size_t i = tree.last_searched_snp + 1; i < tree.snps; ++i) {
        res = compare_snps(tree, tree.read_info, tree.data, &left_child, i, &tree.last_searched_snp);
        if (res > temp) { *result = i; temp = res; }
    }

    printf("%i : %i rest\n ", temp, *result);

//    for (size_t i = 0; i < tree.snps; ++i) 
//        printf("%i ", tree.search_snps[i]);

    printf("\nres: %i, %i\n", right_child.lbound, left_child.node_idx);
    
    //start_index = 1; snp_end = tree.last_searched_snp; haplo_idx = *result; snps = tree.snps; reads = tree.reads;
    //search_helper<<<2, 1>>>(tree, tree.node_manager.nodes, tree.read_info, tree.snp_info, tree.data, &haplo_idx, &snp_end, &start_index, &snps, &reads);
 
    // Now we can start the iterative search 
    
    //size_t a = compare_snps(tree, &left_child, 2);
    tree.last_searched_snp++;
   
   // *result = left_child.alignments;
    
    //free(aligned);
    */
}

}               // End namespace haplo
