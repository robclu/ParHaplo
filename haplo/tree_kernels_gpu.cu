#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include "tree_internal.h"

namespace haplo {

struct BoundsGpu {

bool   lower_is_same;
size_t lower;
size_t upper;
size_t diff;

};
__device__ 
BoundsGpu compare_snps(internal::Tree& tree, const size_t comp_node_idx, const size_t ref_haplo_idx)
{
//    size_t node_idx = node.node_idx;
    size_t read_start = 0, read_end     = 0, row_offset  = 0,
           opp        = 0, same         = 0;
    
    // The bounds for this instance
    BoundsGpu bounds;
            
    // Get a pointer to the node
    const TreeNode* comp_node = &tree.nodes[comp_node_idx];
    
    // Go back up the tree and determine the 
    for (size_t i = tree.last_searched_snp + 1; i > 0; --i) {
      
        // DEBUG  
        printf("Node ID : %i Node HI : %i, Ref HI : %i\n", comp_node->node_idx, comp_node->haplo_idx, ref_haplo_idx);
        
        // Go through the alignments for the node
        for (size_t j = 0; j < comp_node->alignments; ++j) {
            read_start = tree.read_info[comp_node->read_ids[j]].start_index();
            read_end   = tree.read_info[comp_node->read_ids[j]].end_index();
            
            // If any of the comp and ref snps (nodes) have overlapping positions
            if (read_start <= comp_node->haplo_idx && read_start <= ref_haplo_idx &&
                read_end   >= comp_node->haplo_idx && read_end   >= ref_haplo_idx ) {
                row_offset = tree.read_info[comp_node->read_ids[j]].offset();

                uint8_t comp_value = tree.data[row_offset + comp_node->haplo_idx];
                uint8_t ref_value  = tree.data[row_offset + ref_haplo_idx];
                
                if (comp_value == ref_value && ref_value <= 1) {
                    ++same;
                    // DEBUG
                    printf("Adding Same : %i, %i\n", comp_node->read_ids[j], ref_haplo_idx);
                } else if (comp_value != ref_value && ref_value <= 1 && comp_value <=1) {
                    ++opp;
                    // DEBUG
                    printf("Adding Oppp : %i, %i\n", comp_node->read_ids[j], ref_haplo_idx);
                }
            }
        }
        if (i > 1) {
            // Move the pointer up the tree (towards the root)
            const size_t root_idx = comp_node->root_idx;
            comp_node = tree.node_ptr(root_idx);
            
            // DEBUG
            printf("Reassign : Root ID : %i : Node ID :\n", root_idx, comp_node->node_idx); 
        }
    }
    // Format the bounds 
    bounds.lower = min((unsigned int)same, (unsigned int)opp);
    bounds.upper = max((unsigned int)same, (unsigned int)opp);
    bounds.diff  = bounds.upper - bounds.lower;
    bounds.lower_is_same = same <= opp ? true : false;
    
    return bounds;
}

__device__ size_t* realloc(size_t& old_size, size_t new_size, size_t* old)
{
    size_t* new_array = (size_t*)malloc(new_size * sizeof(size_t));

    for (size_t i = 0; i < old_size; i++) new_array[i] = old[i];

    free(old);
    return new_array;
}

// Add any unaligned reads to a node (snp site)
__device__ 
void add_alignments(internal::Tree& tree, const size_t node_idx)
{
    size_t read_offset = 0, align_count = 0;
    
    // Get a pointer to the node
    TreeNode* node = tree.node_ptr(node_idx);
   
    // DEBUG
    printf("AlignS : %i\n", node->haplo_idx);
    
    // Allocate some temp space for the node alignments (this is usually small)
    size_t* alignments = new size_t[tree.snp_info[node->haplo_idx].end_index() - 
                                    tree.snp_info[node->haplo_idx].start_index()];
    size_t* values     = new size_t[tree.snp_info[node->haplo_idx].end_index() - 
                                    tree.snp_info[node->haplo_idx].start_index()];     
    
    // Go through all the unaligned reads
    for (size_t i = tree.last_unaligned_idx; i < tree.reads; ++i) {
        if (i >= tree.snp_info[node->haplo_idx].start_index() &&
            i <= tree.snp_info[node->haplo_idx].end_index()  ) {
           
            // DEBUG
            printf("Test\n");
            
            // If the row crosses the snp
            if (tree.read_info[tree.aligned_reads[i]].start_index() <= node->haplo_idx &&
                tree.read_info[tree.aligned_reads[i]].end_index()   >= node->haplo_idx ) {
            
                // Get the offset in memory of the start of the read
                read_offset  = tree.read_info[i].offset();
                auto element = tree.data[read_offset + node->haplo_idx];
            
                // Do the alignement for the reads to this node
                if ((element == 0 && node->value == 0) || (element == 1 && node->value == 1)) {
                    // DEBUG
                    printf("Align1 : %i\n", i);
                    values[align_count] = 1; alignments[align_count++] = tree.aligned_reads[i];
                } else if ((element == 0 && node->value == 1) || (element == 1 && node->value == 0)) {
                    values[align_count] = 0; alignments[align_count++] = tree.aligned_reads[i];
                    // DEBUG
                    printf("Align2 : %i\n", i);
                }
            }
        }
    }
   
    // Move the found alignments to the node
    node->alignments     = align_count;
    node->read_ids       = new size_t[align_count];
    node->read_values    = new uint8_t[align_count];
    
    for (size_t i = 0; i < align_count; ++i) {
        node->read_ids[i]    = alignments[i];
        node->read_values[i] = (uint8_t)values[i];
    }
    // Clean memory
    free(alignments); free(values);
}


__global__ void search_helper(internal::Tree tree, TreeNode* nodes, ReadInfo* read_info, SnpInfoGpu* snp_info, uint8_t* data, 
        size_t* haplo_idx, size_t* last_snp, size_t* start_idx, size_t* snps, size_t* reads)
{
    // Set node parameters
    BoundsGpu bounds;
    size_t      node_idx  = threadIdx.x + *start_idx;
    TreeNode*   node      = &nodes[node_idx];
    node->haplo_idx         = *haplo_idx;
    
    //size_t res = compare_snps(read_info, data, &node, 1, last_snp, &bounds);
   // size_t temp = 0, res = 0;
  //  for (size_t i = *last_snp + 1; i < *snps; ++i) {
  //      res = compare_snps(tree, read_info, data, node, 2, *last_snp, &bounds);
  //      if (res > temp) { node->haplo_idx = i; temp = res; }
   // }
    
    //add_alignments(tree, read_info, *reads, snp_info, tree.data, node , set_alignments, last_aligned);
}

__device__ size_t start_node_index = 1;                     // For each level, this is the index in the 
                                                            // node array of the first element in the level 
__device__ size_t nodes_in_level = 2;                       // The number of nodes (sub-branches) in the level

__global__ void search_tree(internal::Tree tree, size_t start_ubound, size_t device_index)
{
    // DEBUG 
    printf("Device Index : %i\n", device_index);
    printf("Start Bound  : %i\n" , start_ubound);
    
    struct hipDeviceProp_t device_properties;                // So that we can know the max number of threads

    // Get the properties of the device 
    //
    //hipError_t status = 
    // ---------------------------------------- ROOT NODE -------------------------------------------------

    // DEBUG
    for (size_t i = 0; i < tree.reads; ++i) printf("%i\n", tree.aligned_reads[i]);
    for (size_t i = 0; i < tree.snps;  ++i) printf("%i\n", tree.search_snps[i]);
    
    TreeNode& node = tree.nodes[0];
    node.haplo_idx = tree.search_snps[tree.last_searched_snp];
    node.node_idx  = 0; node.value  = 0;
    
    // Set the alignments for the tree root
    add_alignments(tree, 0);
    
    // DEBUG 
    printf("AlignF : %i\n", node.alignments);

    // Add the alignments to the overall alignments
    for (size_t i = tree.last_unaligned_idx; i < tree.last_unaligned_idx + node.alignments; ++i) {
        tree.aligned_reads[node.read_ids[i - tree.last_unaligned_idx]] = tree.aligned_reads[i];
        tree.aligned_reads[i] = node.read_ids[i - tree.last_unaligned_idx];
    } tree.last_unaligned_idx += node.alignments;
    
    // DEBUG 
    printf("AlignL : %i\n", tree.last_unaligned_idx);
    for (size_t i = 0; i < tree.last_unaligned_idx; ++i) printf("%i ", tree.aligned_reads[i]);
    printf("\nLast Searched: %i\n", tree.last_searched_snp);    

    // Go over all the nodes that have not been searched and see how correlated they are
    size_t max = 0, index = 0; BoundsGpu bounds_temp, bounds_final;
    for (size_t i = tree.last_searched_snp + 1; i < tree.snps; ++i) {
        bounds_temp = compare_snps(tree, node.node_idx, i);
        if (bounds_temp.diff > max) { 
            index        = i; 
            max          = bounds_temp.diff; 
            bounds_final = bounds_temp;
        }
        printf("Result : %i\n", bounds_temp.diff);
        printf("Max    : %i\n", max   );
        printf("Index  : %i\n", index );
   }
  
    // DEBUG
    printf("Most Correlated : %i Correlation : %i\n", index, max);
    
    // The first node has now been searched
    tree.last_searched_snp++;
    
/*    
    // Make the next 2 nodes point back to this one
    TreeNode& left_child = tree.nodes[1]; TreeNode& right_child = tree.nodes[2];
    left_child.root_idx  = 0; right_child.root_idx  = 0;  
    left_child.value     = 1; right_child.value     = 1;
    left_child.node_idx  = 1; right_child.node_idx  = 2;
        
    printf("%i rest\n ",*result);
    
    left_child.haplo_idx = 1; right_child.haplo_idx  = 1;
    printf("%i rest\n ", left_child.node_idx);
    
    add_alignments(tree, tree.read_info, tree.reads, tree.snp_info, tree.data, &left_child , aligned, tree.last_aligned);
    add_alignments(tree, tree.read_info, tree.reads, tree.snp_info, tree.data, &right_child, aligned, tree.last_aligned);
   
    printf("%i rest\n ", left_child.node_idx);

    //left_child.lbound    = bounds.same; left_child.ubound  = *ubound - (bounds.same + left_child.alignments);
    //right_child.lbound   = bounds.opps; right_child.ubound = *ubound - (bounds.opps + right_child.alignments);
    
    // Add the alignments due to these nodes -- same if we use right or left child
    for (size_t i = tree.last_aligned; i < tree.last_aligned + left_child.alignments; ++i) {
            aligned[left_child.read_ids[i - tree.last_aligned]] = aligned[i];
            aligned[i] = left_child.read_ids[i - tree.last_aligned];
    } tree.last_aligned += left_child.alignments;

    printf("%i rest\n ", left_child.node_idx);

    temp = 0;
    for (size_t i = tree.last_searched_snp + 1; i < tree.snps; ++i) {
        res = compare_snps(tree, tree.read_info, tree.data, &left_child, i, &tree.last_searched_snp);
        if (res > temp) { *result = i; temp = res; }
    }

    printf("%i : %i rest\n ", temp, *result);

//    for (size_t i = 0; i < tree.snps; ++i) 
//        printf("%i ", tree.search_snps[i]);

    printf("\nres: %i, %i\n", right_child.lbound, left_child.node_idx);
    
    //start_index = 1; snp_end = tree.last_searched_snp; haplo_idx = *result; snps = tree.snps; reads = tree.reads;
    //search_helper<<<2, 1>>>(tree, tree.node_manager.nodes, tree.read_info, tree.snp_info, tree.data, &haplo_idx, &snp_end, &start_index, &snps, &reads);
 
    // Now we can start the iterative search 
    
    //size_t a = compare_snps(tree, &left_child, 2);
    tree.last_searched_snp++;
   
   // *result = left_child.alignments;
    
    //free(aligned);
    */
}

}               // End namespace haplo
